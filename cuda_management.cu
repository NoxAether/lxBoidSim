#include "hip/hip_runtime.h"
#include "boid_core.h"
#include "defs.h"
#include "grid_cell.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
#include <raylib.h>
}

// Kernel declarations
extern __global__ void build_grid_kernel(Boid *boids, GridCell *grid, int n,
                                         float grid_min_x, float grid_min_y,
                                         float grid_cell_size, int grid_size_x);
extern __global__ void
collision_cohesion_kernel_optimized(Boid *boids, GridCell *grid, int n,
                                    float grid_min_x, float grid_min_y,
                                    float grid_cell_size, int grid_size_x);
extern __global__ void apply_bias_kernel(Boid *boids, int n);
extern __global__ void limit_speed_kernel(Boid *boids, int n);
extern __global__ void update_position_kernel(Boid *boids, int n);
extern __global__ void check_bounds_kernel(Boid *boids, int n, int left_margin,
                                           int right_margin, int top_margin,
                                           int bottom_margin);

// Grid parameters
const float GRID_MIN_X = -150.0f;
const float GRID_MIN_Y = 90.0f;
const int GRID_CELL_COUNT = GRID_SIZE_X * GRID_SIZE_Y;

Boid *createBoidArrayCUDA() {
    Boid *d_boids;
    hipError_t err = hipMalloc(&d_boids, BOID_COUNT * sizeof(Boid));
    if (err != hipSuccess) {
        printf("CUDA malloc error: %s\n", hipGetErrorString(err));
        return NULL;
    }

    // Create temporary host array to initialize device memory
    Boid *h_boids = (Boid *)malloc(BOID_COUNT * sizeof(Boid));
    if (!h_boids) {
        printf("Host boid array creation unsuccessful\n");
        hipFree(d_boids);
        return NULL;
    }

    for (int i = 0; i < BOID_COUNT; i++) {
        Boid *bptr = &h_boids[i];

        // Initialize values
        float angle = ((float)rand() / (float)RAND_MAX) * 2.0f * PI;
        float speed = MIN_SPEED + ((float)rand() / (float)RAND_MAX) *
                                      (MAX_SPEED - MIN_SPEED);

        bptr->x_pos = (float)GetRandomValue(0, SCREEN_WIDTH);
        bptr->y_pos = (float)GetRandomValue(0, SCREEN_HEIGHT);
        bptr->x_vel = cosf(angle) * speed;
        bptr->y_vel = sinf(angle) * speed;
        bptr->close_dx = 0;
        bptr->close_dy = 0;
        bptr->xpos_avg = 0;
        bptr->ypos_avg = 0;
        bptr->neighboring_boids = 0;
        bptr->biasval = DEFAULT_BIAS;
        bptr->is_scout_group_1 = GetRandomValue(0, 1);
        bptr->z_pos = GetRandomValue(-5, 5);
        bptr->type = BOID_TYPE_BASIC; // Default type
    }

    // Copy initialized data to device
    err = hipMemcpy(d_boids, h_boids, BOID_COUNT * sizeof(Boid),
                     hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA memcpy error: %s\n", hipGetErrorString(err));
        hipFree(d_boids);
        free(h_boids);
        return NULL;
    }

    free(h_boids);

    printf("CUDA boid array creation successful\n");
    return d_boids;
}

void freeBoidArrayCUDA(Boid *d_boids) {
    hipFree(d_boids);
    fprintf(stdout, "CUDA boid array freed\n");
}

void runCUDAKernels(Boid *d_boids) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (BOID_COUNT + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate grid memory
    GridCell *d_grid;
    hipMalloc(&d_grid, GRID_CELL_COUNT * sizeof(GridCell));

    // Clear grid
    hipMemset(d_grid, 0, GRID_CELL_COUNT * sizeof(GridCell));

    // Build spatial grid
    build_grid_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_boids, d_grid, BOID_COUNT, GRID_MIN_X, GRID_MIN_Y, GRID_CELL_SIZE,
        GRID_SIZE_X);

    // Run optimized collision/cohesion kernel
    collision_cohesion_kernel_optimized<<<blocksPerGrid, threadsPerBlock>>>(
        d_boids, d_grid, BOID_COUNT, GRID_MIN_X, GRID_MIN_Y, GRID_CELL_SIZE,
        GRID_SIZE_X);

    // Run other kernels
    apply_bias_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_boids, BOID_COUNT);
    limit_speed_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_boids, BOID_COUNT);
    update_position_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_boids,
                                                               BOID_COUNT);
    check_bounds_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_boids, BOID_COUNT,
                                                            -150, 150, 90, 260);

    // Free grid memory
    hipFree(d_grid);

    // Synchronize to check for errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Kernel error: %s\n", hipGetErrorString(err));
    }
}
