#include "hip/hip_runtime.h"
#include "boid_core.h"
#include "defs.h"
#include "grid_cell.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <math.h>

__constant__ float c_visible_range_squared = VISIBLE_RANGE_SQUARED;
__constant__ float c_protected_range_squared = PROTECTED_RANGE_SQUARED;
__constant__ float c_centering_factor = CENTERING_FACTOR;
__constant__ float c_avoid_factor = AVOID_FACTOR;
__constant__ float c_matching_factor = MATCHING_FACTOR;
__constant__ float c_turn_factor = TURN_FACTOR;
__constant__ float c_min_speed = MIN_SPEED;
__constant__ float c_max_speed = MAX_SPEED;
__constant__ float c_max_bias = MAX_BIAS;
__constant__ float c_bias_increment = BIAS_INCREMENT;
__constant__ float c_delta_time = DELTA_TIME;
__constant__ float c_speed_scale = SPEED_SCALE;

// Kernel to build spatial grid
__global__ void build_grid_kernel(Boid *boids, GridCell *grid, int n,
                                  float grid_min_x, float grid_min_y,
                                  float grid_cell_size, int grid_size_x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *b = &boids[i];

    // Calculate grid cell coordinates
    int cell_x = floorf((b->x_pos - grid_min_x) / grid_cell_size);
    int cell_y = floorf((b->y_pos - grid_min_y) / grid_cell_size);

    cell_x = max(0, min(grid_size_x - 1, cell_x));
    cell_y = max(0, min(GRID_SIZE_Y - 1, cell_y));

    int cell_idx = cell_y * grid_size_x + cell_x;

    // Atomically add boid to grid cell
    int pos = atomicAdd(&grid[cell_idx].count, 1);
    if (pos < MAX_BOIDS_PER_CELL) {
        grid[cell_idx].boid_indices[pos] = i;
    }
}

// Optimized collision and cohesion kernel using spatial grid
__global__ void collision_cohesion_kernel_optimized(Boid *boids, GridCell *grid,
                                                    int n, float grid_min_x,
                                                    float grid_min_y,
                                                    float grid_cell_size,
                                                    int grid_size_x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *bptri = &boids[i];

    // Initialize accumulation variables
    float xpos_avg = 0.0f;
    float ypos_avg = 0.0f;
    float xvel_avg = 0.0f;
    float yvel_avg = 0.0f;
    float close_dx = 0.0f;
    float close_dy = 0.0f;
    int neighboring_boids = 0;

    // Calculate which grid cell this boid is in
    int cell_x = floorf((bptri->x_pos - grid_min_x) / grid_cell_size);
    int cell_y = floorf((bptri->y_pos - grid_min_y) / grid_cell_size);

    // Check neighboring cells (3x3 area around the boid)
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            int check_x = cell_x + dx;
            int check_y = cell_y + dy;

            if (check_x < 0 || check_x >= grid_size_x || check_y < 0 ||
                check_y >= GRID_SIZE_Y) {
                continue;
            }

            int cell_idx = check_y * grid_size_x + check_x;
            GridCell *cell = &grid[cell_idx];

            // Check all boids in this cell
            for (int j = 0; j < cell->count; j++) {
                int other_idx = cell->boid_indices[j];
                if (other_idx == i)
                    continue; // Skip self

                Boid *bptrj = &boids[other_idx];
                float dx = bptri->x_pos - bptrj->x_pos;
                float dy = bptri->y_pos - bptrj->y_pos;
                float distance_squared = dx * dx + dy * dy;

                // Collision avoidance
                if (distance_squared < c_protected_range_squared) {
                    close_dx += dx;
                    close_dy += dy;
                }

                // Cohesion and alignment
                if (distance_squared < c_visible_range_squared &&
                    distance_squared > c_protected_range_squared) {
                    xpos_avg += bptrj->x_pos;
                    ypos_avg += bptrj->y_pos;
                    xvel_avg += bptrj->x_vel;
                    yvel_avg += bptrj->y_vel;
                    neighboring_boids++;
                }
            }
        }
    }

    // Apply collision avoidance
    bptri->x_vel += close_dx * c_avoid_factor;
    bptri->y_vel += close_dy * c_avoid_factor;

    // Apply cohesion and alignment
    if (neighboring_boids > 0) {
        xpos_avg /= neighboring_boids;
        ypos_avg /= neighboring_boids;
        xvel_avg /= neighboring_boids;
        yvel_avg /= neighboring_boids;

        bptri->x_vel += (xpos_avg - bptri->x_pos) * c_centering_factor;
        bptri->y_vel += (ypos_avg - bptri->y_pos) * c_centering_factor;

        bptri->x_vel += (xvel_avg - bptri->x_vel) * c_matching_factor;
        bptri->y_vel += (yvel_avg - bptri->y_vel) * c_matching_factor;
    }
}

// Kernel for applying bias
__global__ void apply_bias_kernel(Boid *boids, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *bptr = &boids[i];

    if (bptr->is_scout_group_1) {
        if (bptr->x_vel > 0)
            bptr->biasval = fminf(c_max_bias, bptr->biasval + c_bias_increment);
        else
            bptr->biasval =
                fmaxf(c_bias_increment, bptr->biasval - c_bias_increment);

        bptr->x_vel =
            (1.0f - bptr->biasval) * bptr->x_vel + bptr->biasval * 1.0f;
    } else {
        if (bptr->x_vel < 0)
            bptr->biasval = fminf(c_max_bias, bptr->biasval + c_bias_increment);
        else
            bptr->biasval =
                fmaxf(c_bias_increment, bptr->biasval - c_bias_increment);

        bptr->x_vel =
            (1.0f - bptr->biasval) * bptr->x_vel - bptr->biasval * 1.0f;
    }
}

// Kernel for speed limiting
__global__ void limit_speed_kernel(Boid *boids, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *bptr = &boids[i];
    float speed = sqrtf(bptr->x_vel * bptr->x_vel + bptr->y_vel * bptr->y_vel);

    if (speed < c_min_speed) {
        bptr->x_vel = (bptr->x_vel / speed) * c_min_speed;
        bptr->y_vel = (bptr->y_vel / speed) * c_min_speed;
    }
    if (speed > c_max_speed) {
        bptr->x_vel = (bptr->x_vel / speed) * c_max_speed;
        bptr->y_vel = (bptr->y_vel / speed) * c_max_speed;
    }
}

// Kernel for updating positions
__global__ void update_position_kernel(Boid *boids, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *bptr = &boids[i];
    bptr->x_pos += bptr->x_vel * c_delta_time * c_speed_scale;
    bptr->y_pos += bptr->y_vel * c_delta_time * c_speed_scale;
}

// Kernel for boundary checking
__global__ void check_bounds_kernel(Boid *boids, int n, int left_margin,
                                    int right_margin, int top_margin,
                                    int bottom_margin) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n)
        return;

    Boid *bptr = &boids[i];

    if (bptr->x_pos < left_margin)
        bptr->x_vel += c_turn_factor;

    if (bptr->x_pos > right_margin)
        bptr->x_vel -= c_turn_factor;

    if (bptr->y_pos > bottom_margin)
        bptr->y_vel -= c_turn_factor;

    if (bptr->y_pos < top_margin)
        bptr->y_vel += c_turn_factor;
}
