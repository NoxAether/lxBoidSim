#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Include CUDA headers first
#include <hip/hip_runtime.h>

// Then include other headers
#include <raylib.h>
#include <rcamera.h>

#include "boid_core.h"
#include "controls.h"
#include "defs.h"
#include "drawing.h"
#include "player.h"
#include "world.h"

// Function declarations for CUDA operations
Boid *createBoidArrayCUDA();
void freeBoidArrayCUDA(Boid *d_boids);
void runCUDAKernels(Boid *d_boids);

int main(void) {
    srand((unsigned)time(NULL));
    printf("CUDA Boids Simulation...\n");

    // Initialize CUDA device boid array
    Boid *d_boids = createBoidArrayCUDA();
    fprintf(stdout, "CUDA boid array setup and initialized\n");

    // Raylib setup
    SetTargetFPS(FPS);
    InitWindow(SCREEN_WIDTH, SCREEN_HEIGHT, "CUDA Boids Simulation");

    Player player = core_camera();
    DisableCursor();

    // Host copy for rendering
    Boid *h_boids = (Boid *)malloc(BOID_COUNT * sizeof(Boid));
    if (!h_boids) {
        printf("Host boid array allocation failed\n");
        return 1;
    }

    while (!WindowShouldClose()) {
        // Run CUDA kernels for simulation
        runCUDAKernels(d_boids);

        // Copy results back to host for rendering
        hipError_t err =
            hipMemcpy(h_boids, d_boids, BOID_COUNT * sizeof(Boid),
                       hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }

        core_input(&player);
        UpdateCamera(&player.camera, player.camera_mode);

        // Drawing
        BeginDrawing();
        ClearBackground(BLACK);
        BeginMode3D(player.camera);

        draw_world();
        draw_all(h_boids);
        draw_player(player);

        EndMode3D();
        DrawFPS(10, 10);
        EndDrawing();
    }

    // Cleanup
    freeBoidArrayCUDA(d_boids);
    free(h_boids);

    fprintf(stdout, "CUDA memory cleaned up\n");
    CloseWindow();

    return 0;
}
